

#include <hip/hip_runtime.h>
#include<stdio.h>
#define BLOCK 1024
#define N 100000

__global__ void compress(int * d_in, int * d_start, int * d_end){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){ 
    if (d_in[idx] < d_in[idx+1])
      d_start[idx] = idx+2;       

    if (idx == 0 and d_in[idx] == 1)
      d_start[idx] = 1;
                          
    if (idx == N and d_in[idx] == 1)
      d_end[idx] = N;
  
    if (d_in[idx] > d_in[idx+1])
      d_end[idx] = idx+1; 
  } 
}

__global__ void compressleft(int * d_start, int * d_end ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){ 
    if (d_start[idx] == 0 && d_start[idx+1] != 0 )  {
      d_start[idx]=d_start[idx+1]; d_start[idx+1]=0;
    }

    if (d_end[idx] == 0 && d_end[idx+1] != 0 )  {
      d_end[idx]=d_end[idx+1]; d_end[idx+1]=0;
    }
  }
}

__global__ void compressreduce(int * d_start, int * d_end, int * d_out){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N){ 
    if (idx ==0){
        d_out[idx] = d_start[idx];
        d_out[idx+1] = d_end[idx]-d_start[idx]+1;
    }
    else{
      if (d_start[idx]!=0 && d_end[idx]!=0){
        d_out[idx*2] = d_start[idx];
        d_out[idx*2+1] = d_end[idx]-d_start[idx]+1;
      }
    }
  }
}

__global__ void decompress(int * d_in, int * d_out){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    d_out[idx] = 0 ;
    if (idx % 2 == 0){
        for (int i=d_in[idx]-1; i<d_in[idx] + d_in[idx+1] -1; i++){
            d_out[i] = 1;
        }
    }
  }
  __syncthreads();
}

int main(){
  /* initialise */
  int *d_in, *d_out, *d_start, *d_end;
  int *h_in, *h_out, *h_start, *h_end;

  h_in = (int *)malloc(sizeof(int)*N);
  h_out = (int *)malloc(sizeof(int)*N);
  h_start = (int *)malloc(sizeof(int)*N);
  h_end = (int *)malloc(sizeof(int)*N);

  for(int a = 0; a < N ; a++){
    h_in[a] = rand() % 2 ;}

  hipMalloc((void**) &d_in, sizeof(int)*N);
  hipMalloc((void**) &d_out, sizeof(int)*N);
  hipMalloc((void**) &d_start, sizeof(int)*N);
  hipMalloc((void**) &d_end, sizeof(int)*N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float ms = 0;
  float total = 0 ;

  
  //printf("input:   \t");
  for (int i = 0 ; i<N; i++){
    //printf("%d ",h_in[i]);
  }

  int n_blocks = (N/1024) + 1 ; 
/////////////////////* compress1 kernel call *///////////////////////////
  hipMemcpy(d_in ,h_in, N*sizeof(int), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  compress<<<n_blocks,1024>>>(d_in, d_start, d_end);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  total = total + ms; 

  hipMemcpy(h_start ,d_start, N*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_end ,d_end, N*sizeof(int), hipMemcpyDeviceToHost);


  /// intermediate print for understanding  
  /*
  printf("\ncompress1 start:");
  for (int i = 0 ; i<N; i++){
    if (h_start[i] !=0)
        printf(" %d",h_start[i]);
  }
  printf("\ncompress1 end: \t");
  for (int i = 0 ; i<N; i++){
    if (h_end[i] !=0)
        printf(" %d",h_end[i]);
  }
  */
 
 ///////////////* compress2 left kernel call */ /////////////////////////
  for (int i = 0;i<N;i++){
    hipEventRecord(start);
    compressleft<<<n_blocks,1024>>>(d_start, d_end);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    total = total + ms; 

    hipDeviceSynchronize();
  }
  hipMemcpy(h_start ,d_start, N*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_end ,d_end, N*sizeof(int), hipMemcpyDeviceToHost);

  /// intermediate print for understanding   
  /*
  printf("\ncompress2 start:");
  for (int i = 0 ; i<N; i++){
    if (h_start[i] !=0)
        printf(" %d",h_start[i]);
  }
  printf("\ncompress2 end: \t");
  for (int i = 0 ; i<N; i++){
    if (h_end[i] !=0)
        printf(" %d",h_end[i]);
  }
  */
/////////////////////* compress reduce kernel call *///////////////////////////
  
  hipEventRecord(start);
  compressreduce<<<n_blocks,1024>>>(d_start, d_end, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  total = total + ms; 

  hipMemcpy(h_out ,d_out, N*sizeof(int), hipMemcpyDeviceToHost);

  //printf("\n\ncompress:\t");
  for (int i = 0 ; i<N; i++){
    //if (h_out[i] !=0)
      //printf(" %d",h_out[i]);
  }

///////////////////* decompress kernel call */////////////////////////
  hipMemcpy(d_in ,h_out, N*sizeof(int), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  decompress<<<n_blocks,1024>>>(d_in, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  total = total + ms; 

  hipMemcpy(h_out, d_out, N*sizeof(int), hipMemcpyDeviceToHost);
  
  //printf("\n\ndecompress: \t");
  for (int i = 0 ; i<N; i++){
    //printf(" %d",h_out[i]);
  }

  printf("\n\nsize of N %d\n", N);
  printf("\ntotal GPU time taken %lf\n",total/1000) ;

return 0;
}

/*

nvcc hw4_25.cu -o hw4_25cu.out 
./hw4_25cu.out 

*/