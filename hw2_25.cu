

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void varcheck(int *x, int *y, int *z){
	int tid=threadIdx.x;
    
    if (tid %2 ==0)
        if (x==y)
            x=z;

    if (tid %2 ==1)
        if (x==z)
            x=y;
}

int main(){
  /* initialise */
  int *d_x, *d_y, *d_z ;
  int *h_x, *h_y, *h_z ;

  hipMalloc((void**) &d_x, sizeof(int));
  hipMalloc((void**) &d_y, sizeof(int));
  hipMalloc((void**) &d_z, sizeof(int));

  //////

  //h_x = (int *)malloc(sizeof(char)*1);
  //h_y = (int *)malloc(sizeof(char)*1);
  //h_z = (int *)malloc(sizeof(char)*1);

  //h_x[1]= 15; h_y[1]= 15 ; h_z[1]= 26 ;

  //////

  hipMemcpy(d_x, h_x, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_z, h_z, sizeof(int), hipMemcpyHostToDevice);

  varcheck<<<1,2>>>(d_x,d_y,d_z);
  hipMemcpy(h_x, d_x, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\n%d",&h_x);
	return 0;
}
