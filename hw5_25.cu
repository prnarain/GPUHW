

#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 1024
#define BLOCK 1024

__global__ void reduceV1(int *elems){
	int id,i;

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(i=N/2; i; i/=2) {
		if(id<i)
			elems[id] += elems[id+i];
		__syncthreads();
	}
	if(id==0)
		printf("GPU Sum approach 1 is %d %d\n",id , elems[0]);
}

__global__ void reduceV2(int *elems){
	int id, i;

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(i=N/2; i; i/=2) {
		if(id<i){
			if (id%2 ==0){
				elems[id] += elems[id+i];
			}
			if (id%2 ==1){
				elems[id] += elems[id+i];
			}
		}
		__syncthreads();
	}
	if(id==0)
		printf("GPU Sum approach 2 is %d %d\n",id , elems[0]);
}

__global__ void reduceV3(int *elems){
	int id, i;

	id=threadIdx.x+blockIdx.x*blockDim.x;
	for(i=N/2; i; i/=2) {
		if(id<i){
			if (id<i/2){
				elems[id] += elems[id+i];
			}
			if (id>=i/2 ){
				elems[id] += elems[id+i];
			}
		}
		__syncthreads();
	}
	if(id==0)
		printf("GPU Sum approach 3 is %d %d\n",id , elems[0]);
}


int main(){
	int host[N],i;
	int sum=0;

	for(i=0;i<N;i++){
		host[i]=rand()%20;
		sum+=host[i];
	}	

	printf("CPU Sum is %d\n",sum);

	int *d_elems, *d_out;

	hipMalloc(&d_elems,N*sizeof(int));
	hipMalloc(&d_out,N*sizeof(int));

	hipMemcpy(d_elems,host,N*sizeof(int),hipMemcpyHostToDevice);
	reduceV1<<<(N+BLOCK-1)/BLOCK,BLOCK>>>(d_elems);
	hipDeviceSynchronize();

	hipMemcpy(d_elems,host,N*sizeof(int),hipMemcpyHostToDevice);
	reduceV2<<<(N+BLOCK-1)/BLOCK,BLOCK>>>(d_elems);
	hipDeviceSynchronize();

	hipMemcpy(d_elems,host,N*sizeof(int),hipMemcpyHostToDevice);
	reduceV3<<<(N+BLOCK-1)/BLOCK,BLOCK>>>(d_elems);
	hipDeviceSynchronize();

	return 0;
}

/*

nvcc hw5_25.cu -o hw5_25cu.out 

*/