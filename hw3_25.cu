

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 1000

__global__ void encrypt(char *d_in, char *d_out){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int out; 
    if (i<N)
    {
        out = d_in[i] + (i+1) + 1 ; 
        if (out > 122) 
        out = (out - 123)%26 + 97 ; 
        d_out[i] = out;
    }
    __syncthreads();
}

__global__ void decrypt(char *d_in, char *d_out){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int out; 
    if (i<N)
    {
        out = d_in[i] - (i+1) - 1 ;
        if (out  < 97) 
            out = 122 - (96-out)%26   ; 
        d_out[i] = out;
    }
    __syncthreads();
}

int main(){
    
  /* initialise */
  char *d_in, *d_out;
  char *h_in, *h_out, *h_outf;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float ms = 0;
  float total = 0 ; 

  h_in = (char *)malloc(sizeof(char)*N);
  h_out = (char *)malloc(sizeof(char)*N);
  h_outf = (char *)malloc(sizeof(char)*N);
  
  for(int a = 0; a < N ; a++){
    h_in[a] = "abcdefghijklmnopqrstuvwxyz"[random () % 26] ;}

  hipMalloc((void**) &d_in, sizeof(char)*N);
  hipMalloc((void**) &d_out, sizeof(char)*N);

  int n_blocks = N/1024 + 1 ; 
  
  hipMemcpy(d_in, h_in, N*sizeof(char), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  encrypt<<<n_blocks,N>>>(d_in, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  total = total + ms; 

  hipMemcpy(h_out, d_out, N*sizeof(char), hipMemcpyDeviceToHost);
  
  hipMemcpy(d_in, h_out, N*sizeof(char), hipMemcpyHostToDevice);
  
  hipEventRecord(start);
  decrypt<<<n_blocks,N>>>(d_in, d_out);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&ms, start, stop);
  total = total + ms; 

  hipMemcpy(h_outf, d_out, N*sizeof(char), hipMemcpyDeviceToHost);

  //printf("IP \t ENC \t DEC \t\t",N);
  //printf("\n");
  for(int a = 0; a < N ; a++){
    //printf("\n%d \t %d \t %d ",h_in[a], h_out[a], h_outf[a]) ;
    //printf("\n%c \t %c \t %c ",h_in[a], h_out[a], h_outf[a]) ;
  }
  
  printf("\n");
  printf("size of N %d\n", N);
  printf("\ntotal GPU time taken %lf\n",total) ;

  return 0;
}
